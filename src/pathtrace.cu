#include "hip/hip_runtime.h"
#include "pathtrace.h"

#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>
#include <iostream>
#include <thrust/random.h>
#include <thrust/sort.h>
#include <thrust/count.h>
#include <thrust/execution_policy.h>
#include <thrust/remove.h>

#include "bsdf.h"
#include "sceneStructs.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include "utilities.h"
#include "intersections.h"
#include "interactions.h"
#include "common.h"
#include "pathtraceImpl.h"
#include "mesh.h"

#define SORT_RAYS 1

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution, int iter, glm::vec3* image)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < resolution.x && y < resolution.y)
	{
		int index = x + (y * resolution.x);
		glm::vec3 pix = image[index];

		glm::ivec3 color;
		color.x = glm::clamp((int)(pix.x / iter * 255.0), 0, 255);
		color.y = glm::clamp((int)(pix.y / iter * 255.0), 0, 255);
		color.z = glm::clamp((int)(pix.z / iter * 255.0), 0, 255);

		// Each thread writes one pixel location in the texture (textel)
		pbo[index].w = 0;
		pbo[index].x = color.x;
		pbo[index].y = color.y;
		pbo[index].z = color.z;
	}
}

static Scene* hst_scene = NULL;
static GuiDataContainer* guiData = NULL;
static glm::vec3* dev_image = NULL;
static Geom* dev_geoms = NULL;
int lightCount = 0;
static Geom* device_light_geoms = nullptr;
static Material* dev_materials = NULL;
static PathSegment* dev_paths = NULL;
static ShadeableIntersection* dev_path_intersections = NULL;
static int* device_path_matIds = nullptr;
static int* device_pathAlive = nullptr;
static StaticMeshData_Device host_object_staticMeshData;
static StaticMeshData_Device* device_staticMeshData;
static StaticMeshData_Host staticMeshData;
std::string MeshPath = "../models/cube.obj";


void InitDataContainer(GuiDataContainer* imGuiData)
{
	guiData = imGuiData;
}

void pathtraceNewFrame(Scene* scene)
{
	const Camera& cam = hst_scene->state.camera;
	const int pixelcount = cam.resolution.x * cam.resolution.y;
	hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));
	hipMemset(dev_path_intersections, 0, pixelcount * sizeof(ShadeableIntersection));
	hipMemset(device_path_matIds, -1, pixelcount * sizeof(int));
	hipMemset(device_pathAlive, -1, pixelcount * sizeof(int));
}

void pathtraceCreate(Scene* scene)
{
	hst_scene = scene;

	const Camera& cam = hst_scene->state.camera;
	const int pixelcount = cam.resolution.x * cam.resolution.y;

	hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
	hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));

	hipMalloc(&dev_paths, pixelcount * sizeof(PathSegment));

	hipMalloc(&dev_geoms, scene->geoms.size() * sizeof(Geom));
	hipMemcpy(dev_geoms, scene->geoms.data(), scene->geoms.size() * sizeof(Geom), hipMemcpyHostToDevice);

	{
		// populate device_light_geoms
		std::vector<Geom> light_geoms;
		for (int i = 0; i < scene->geoms.size(); i++)
		{
			if (scene->materials[scene->geoms[i].materialid].emittance > 0.)
			{
				light_geoms.push_back(scene->geoms[i]);
			}
		}
		lightCount = light_geoms.size();
		std::cout << "number of light geoms: " << lightCount << std::endl;
		hipMalloc(&device_light_geoms, lightCount * sizeof(Geom));
		hipMemcpy(device_light_geoms, light_geoms.data(), lightCount * sizeof(Geom), hipMemcpyHostToDevice);
	}

	hipMalloc(&dev_materials, scene->materials.size() * sizeof(Material));
	hipMemcpy(dev_materials, scene->materials.data(), scene->materials.size() * sizeof(Material),
	           hipMemcpyHostToDevice);

	hipMalloc(&dev_path_intersections, pixelcount * sizeof(ShadeableIntersection));
	hipMemset(dev_path_intersections, 0, pixelcount * sizeof(ShadeableIntersection));
	// allocate device memory for materialIds
	hipMalloc(&device_path_matIds, pixelcount * sizeof(int));
	hipMemset(device_path_matIds, -1, pixelcount * sizeof(int));

	hipMalloc(&device_pathAlive, pixelcount * sizeof(int));
	hipMemset(device_pathAlive, -1, pixelcount * sizeof(int));
	checkCUDAError("pathtraceInit");

	ReadObjMesh(staticMeshData, MeshPath);
	CreateDeviceObject(&device_staticMeshData, host_object_staticMeshData, staticMeshData);
}

void pathtraceFree()
{
	hipFree(dev_image); // no-op if dev_image is null
	hipFree(dev_paths);
	hipFree(dev_geoms);
	hipFree(dev_materials);
	hipFree(dev_path_intersections);
	checkCUDAError("pathtraceFree");
}

// Add the current iteration's output to the overall image
__global__ void finalGather(int nPaths, glm::vec3* image, PathSegment* iterationPaths)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index < nPaths)
	{
		PathSegment iterationPath = iterationPaths[index];
		image[iterationPath.pixelIndex] += iterationPath.Contribution;
	}
}

/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */

void pathtrace(uchar4* pbo, int frame, int iter)
{
	static int* materialIdStart = new int[hst_scene->materials.size()];
	static int* materialIdEnd = new int[hst_scene->materials.size()];
	const int traceDepth = 10; // = hst_scene->state.traceDepth;
	const Camera& cam = hst_scene->state.camera;
	const int pixelcount = cam.resolution.x * cam.resolution.y;

	// 2D block for generating ray from camera
	const dim3 blockSize2d(8, 8);
	const dim3 blocksPerGrid2d(
		(cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
		(cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

	// 1D block for path tracing
	const int blockSize1d = 128;
	generateRayFromCamera<<<blocksPerGrid2d, blockSize2d>>>(cam, iter, traceDepth, dev_paths, device_pathAlive);
	checkCUDAError("generate camera ray");
	int depth = 0;
	PathSegment* dev_path_end = dev_paths + pixelcount;
	PathSegment* dev_path_begin = dev_paths;
	int total_paths = pixelcount;
	int num_paths = pixelcount;
	// --- PathSegment Tracing Stage ---
	// Shoot ray into scene, bounce between objects, push shading chunks
	std::cout << "New Frame" << std::endl;
	for (int i = 0; i < 2; i++)
	{
		// clean shading chunks
		hipMemset(dev_path_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

		dim3 numblocksPathSegmentTracing = (total_paths + blockSize1d - 1) / blockSize1d;
		{
			// tracing
			computeIntersections << <numblocksPathSegmentTracing, blockSize1d >> >(
				depth,
				num_paths,
				dev_path_begin,
				dev_geoms,
				hst_scene->geoms.size(),
				dev_path_intersections,
				device_path_matIds, device_pathAlive, device_staticMeshData
			);
		}
		{
			struct isPathAlive
			{
				__host__ __device__
				bool operator()(const int x)
				{
					return (x % 2) == 0;
				}
			};
			auto end_iter = thrust::remove(thrust::device, device_pathAlive, device_pathAlive + num_paths, -1);
			num_paths = end_iter - device_pathAlive;
//#if SORT_RAYS
//TODO sorting rays by material id
//			thrust::sort_by_key(
//				thrust::device,
//				device_path_matIds,
//				device_path_matIds + num_paths,
//				thrust::make_zip_iterator(
//					thrust::make_tuple(dev_path_intersections, dev_paths, device_pathAlive)
//				));
//#endif
		}
		std::cout << "number of paths: " << num_paths << std::endl;
		numblocksPathSegmentTracing = (total_paths + blockSize1d - 1) / blockSize1d;
		generateRayFromIntersections << <numblocksPathSegmentTracing, blockSize1d >> >(
			iter, num_paths, dev_path_begin,
			dev_path_intersections, dev_materials,
			hst_scene->geoms.size(), dev_geoms, device_light_geoms, device_pathAlive);
		depth++;
		if (guiData != NULL)
		{
			guiData->TracedDepth = depth;
		}
	}

	// Assemble this iteration and apply it to the image
	dim3 numBlocksPixels = (pixelcount + blockSize1d - 1) / blockSize1d;
	finalGather<<<numBlocksPixels, blockSize1d>>>(total_paths, dev_image, dev_paths);

	// Send results to OpenGL buffer for rendering
	sendImageToPBO<<<blocksPerGrid2d, blockSize2d>>>(pbo, cam.resolution, iter, dev_image);

	// Retrieve image from GPU
	hipMemcpy(hst_scene->state.image.data(), dev_image,
	           pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);

	checkCUDAError("pathtrace");
}
