#include "hip/hip_runtime.h"
#include "pathtraceImpl.h"

#include "bsdf.h"
#include "common.h"
#include "geometry.h"
#include "interactions.h"
#include "intersections.h"
#include "sceneStructs.h"
#include "utilities.h"
#include "mesh.h"
#include "material.h"
#include "renderproxy.h"
#include "scene.h"

#define USE_MESH_GRID_ACCELERATION 1

/**
* Generate PathSegments with rays from the camera through the screen into the
* scene, which is the first bounce of rays.
*
* Antialiasing - add rays for sub-pixel sampling
* motion blur - jitter rays "in time"
* lens effect - jitter ray origin positions based on a lens
*/
__global__ void generateRayFromCamera(Camera cam, int iter, int traceDepth, PathSegment* pathSegments, int* dev_pathAlive)
{
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    if (x < cam.resolution.x && y < cam.resolution.y) {
        int index = x + (y * cam.resolution.x);
        dev_pathAlive[index] = index;
        PathSegment& segment = pathSegments[index];
        thrust::default_random_engine rng = makeSeededRandomEngine(iter, index, traceDepth);
        thrust::uniform_real_distribution<float> u01(0, 1);
		float ramdomNumber1 = u01(rng);
		float ramdomNumber2 = u01(rng);
        segment.ray.origin = cam.position;
        segment.Contribution = glm::vec3(0.0f, 0.0f, 0.0f);
		segment.BSDF = glm::vec3(1.0f, 1.0f, 1.0f);
		segment.PDF = 1.0f;
        segment.Cosine = 1.0f;

        // TODO: implement antialiasing by jittering the ray
        segment.ray.direction = glm::normalize(cam.view
            - cam.right * cam.pixelLength.x * ((float)x + ramdomNumber1 - (float)cam.resolution.x * 0.5f)
            - cam.up * cam.pixelLength.y * ((float)y + ramdomNumber2 - (float)cam.resolution.y * 0.5f)
        );

        segment.pixelIndex = index;
        segment.remainingBounces = traceDepth;
		segment.debug = glm::vec3(1, 0, 1);
    }
}

__device__ float getIntersectionGeometryIndex(
    glm::vec3& debug,
    int& hit_geom_index,
    ShadeableIntersection& OutIntersect,
    Ray& InRayWorld,
    int geoms_size,
	Geom* geoms
)
{
    float t = -1.0f;
    float t_min = FLT_MAX;
    bool outside = true;
    glm::vec3 debug1;
    ShadeableIntersection TempIntersect;
    // naive parse through global geoms

    for (int i = 0; i < geoms_size; i++)
    {
        Geom& geom = geoms[i];

        if (geom.type == CUBE)
        {
            t = boxIntersectionTest(geom, InRayWorld, TempIntersect);
        }
        else if (geom.type == SPHERE)
        {
            t = sphereIntersectionTest(geom, InRayWorld, TempIntersect);
        }
    	else if (geom.type == PLANE)
        {
            t = planeIntersectionTest(geom, InRayWorld, TempIntersect);
        }else if (geom.type==MESH && geom.MeshProxy_Device!=nullptr)
        {
#if USE_MESH_GRID_ACCELERATION
			t = meshIntersectionTest_Optimized(debug, geom, geom.MeshProxy_Device, InRayWorld, TempIntersect);
#else
            t = meshIntersectionTest(geom, geom.MeshProxy_Device, InRayWorld, OutIntersect);
#endif
        }
        // Compute the minimum t from the intersection tests to determine what
        // scene geometry object was hit first.
        if (t > 0.0f && t_min > t)
        {
            t_min = t;
            hit_geom_index = i;
			OutIntersect = TempIntersect;
        }
    }
    return t_min;
}

__global__ void computeIntersections(
    int depth,
    int num_paths,
    PathSegment* pathSegments,
    Scene::RenderProxy* scene,
    ShadeableIntersection* intersections,
    int* device_materialIds, int* dev_pathAlive)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int path_index = dev_pathAlive[tid];
    if (path_index < 0 || tid >= num_paths)
    {
        return;
    }
    PathSegment pathSegment = pathSegments[path_index];
	// If there are no remaining bounces, no need to trace
    if (pathSegment.remainingBounces <= 0) {
        intersections[path_index].materialId = -1;
        device_materialIds[path_index] = -1;
        dev_pathAlive[tid] = -1;
        return;
	}
    int hit_geom_index = -1;
    ShadeableIntersection Intersect = intersections[path_index];
    glm::vec3 debug;
    getIntersectionGeometryIndex(debug,
        hit_geom_index, Intersect,
        pathSegment.ray, scene->geoms_size, scene->geoms_Device);

    if (hit_geom_index == -1)
    {
        Intersect.materialId = -1;
        device_materialIds[path_index] = -1;
        dev_pathAlive[tid] = -1;
    }
    else
    {
        // The ray hits something
        int matId = scene->geoms_Device[hit_geom_index].materialid;
        device_materialIds[path_index] = matId;
        Intersect.materialId = matId;
        pathSegment.debug = glm::vec3(Intersect.uv.x, Intersect.uv.y, 0.f);
		pathSegments[path_index] = pathSegment;
    }
	intersections[path_index] = Intersect;
}

__host__ __device__ float power_heuristic(float pdf_a, float pdf_b) {
    float a = pdf_a * pdf_a;
    float b = pdf_b * pdf_b;
    return a / (a + b);
}

//__device__ void bsdfDiffuse(glm::vec3& outBSDF, float& outPDF, const Ray& in_wi, glm::vec3 surfaceNormal, Material* material)
//{
//    float absdot = glm::abs(glm::dot(in_wi.direction, surfaceNormal));
//    float pdf = absdot * INV_PI;
//	outPDF = pdf;
//    outBSDF = material->color * INV_PI;
//}
//
//__device__ void bsdfDiffuseSample(glm::vec3& outBSDF, float& outPDF, Ray& out_wi, glm::vec3 p,
//    glm::vec3 surfaceNormal, Material* material,
//    thrust::default_random_engine& rng)
//{
//    // set segment.ray.origin
//    out_wi.origin = p;
//    // set segment.ray.direction on random position
//    out_wi.direction = calculateRandomDirectionInHemisphere(surfaceNormal, rng);
//    bsdfDiffuse(outBSDF, outPDF, out_wi, surfaceNormal, material);
//}

__device__ void bsdfPBR(glm::vec3& debug, glm::vec3& outBSDF, 
    float& outPDF, glm::vec2 uv,
    const Material& material, 
    glm::vec3 L /*Light direction*/, 
    glm::vec3 V /*View direction*/, 
    glm::vec3 N /*Normal*/)
{
    BRDF_Params params;
    if (material.BaseColorTextureProxy_Device != nullptr)
    {
        params.baseColor = GetColorDevice(*material.BaseColorTextureProxy_Device, uv);
    }
	else
	{
        params.baseColor = material.color;
	}
    debug = params.baseColor;
    params.roughness = material.roughness;
    float absdot = glm::abs(glm::dot(L, N));
    float pdf = absdot * INV_PI;

    glm::vec3 tangentX, tangentY;
    // calculate tangentX and tangentY
    glm::vec3 v(0.f, 1.f, 0.f);
    if (glm::abs(glm::dot(v, N)) > 0.999f)
    {
        v = glm::vec3(1.f, 0.f, 0.f);
    }
    tangentX = glm::normalize(glm::cross(v, N));
    tangentY = glm::normalize(glm::cross(N, tangentX));
    outPDF = pdf;
    outBSDF = BRDF(params, L, V, N, tangentX, tangentY);
}

__device__ void bsdfPBRSample(glm::vec3& debug, glm::vec3& outBSDF, float& outPDF, Ray& out_wi, glm::vec3 ViewDir, glm::vec3 p,
    glm::vec3 surfaceNormal, glm::vec2 uv, const Material& material, 
    thrust::default_random_engine& rng)
{
    // set segment.ray.origin
    out_wi.origin = p;
    // set segment.ray.direction on random position
    out_wi.direction = calculateRandomDirectionInHemisphere(surfaceNormal, rng);
	bsdfPBR(debug, outBSDF, outPDF, uv, material, out_wi.direction, ViewDir, surfaceNormal);
}

__device__ void bsdfEmitting(PathSegment* wo, Material* material)
{
    //wo->remainingBounces = 0;
    //wo->color *= material->emittance;
}

__device__ void getGeomPDF(float& outPdf, Geom& InGeom)
{
    if (InGeom.type == CUBE)
    {
        //pdfCube(outPdf, InGeom);
    }
    else if (InGeom.type == PLANE)
    {
        pdfPlane(outPdf, InGeom);
    }
}

__device__ bool sampleLightFromIntersections(
    glm::vec3 &debug, 
    glm::vec3& outDirectLight,
    float& outPdf,
    Ray& wj,
	glm::vec3 p, // intersection point
    const Material& light_mat,
    Geom& light_geom,
    int geomSize,
	Geom* geoms,
    thrust::default_random_engine& rng
)
{
    glm::vec3 lightPosition;
    glm::vec3 lightNormal;
	outPdf = 1.0f;
    sampleGeometry(light_geom, lightPosition, lightNormal, outPdf, rng);
    wj.origin = p;
	wj.direction = glm::normalize(lightPosition - wj.origin);
    float distance = glm::length(lightPosition - wj.origin);
    // convert pdf from area to solid angle
	// pdf_L(direct light) = pdf_A * (distance * distance) / (n . wj)
	float dotProduct = abs(glm::dot(lightNormal, wj.direction));
    outPdf /= dotProduct;
    outPdf *= (distance * distance);
	int hit_geom_index = -1;
    ShadeableIntersection _;
	getIntersectionGeometryIndex(debug, 
		hit_geom_index,
        _,
        wj, geomSize, geoms);
    outDirectLight += light_mat.emittance;
    return dotProduct > 0.0001f && hit_geom_index == 0;//&& glm::length(tmp_intersect - lightPosition) < 0.001f;
}

__device__ void SampleDirectLightMIS(glm::vec3& debug, glm::vec3& OutContribution, 
    glm::vec3 In_p, glm::vec3 InViewDir, glm::vec3 InSurfaceNormal, Material& InSurfaceMat, 
    Geom& InLightGeom, Material& InLightMat, int GeomSize, Geom* Geoms, 
    thrust::default_random_engine& rng)
{
    glm::vec3 directLight;
    float pdf_Ld;
    glm::vec3 bsdf;
    glm::vec2 uv;
    float pdf_bsdf;
    Ray wj;
    bool sampledDirectLight = sampleLightFromIntersections(debug, directLight, pdf_Ld, wj, In_p, InLightMat, InLightGeom, GeomSize, Geoms, rng);
    if (sampledDirectLight)
    {
        //bsdfDiffuse(bsdf, pdf_bsdf, wj, InSurfaceNormal, &InSurfaceMat);
        bsdfPBR(debug, bsdf, pdf_bsdf, uv, InSurfaceMat, wj.direction, InViewDir, InSurfaceNormal);
        float weight = power_heuristic(pdf_Ld, pdf_bsdf);
        OutContribution += directLight * bsdf * glm::max(0.f, glm::dot(wj.direction, InSurfaceNormal)) * weight / pdf_Ld;
    }
	else
    {
	    OutContribution = glm::vec3(0.f);
		return;
    }
    bsdfPBRSample(debug, bsdf, pdf_bsdf, wj, InViewDir, In_p, InSurfaceNormal, uv, InSurfaceMat, rng);
    //bsdfDiffuseSample(bsdf, pdf_bsdf, wj, In_p, InSurfaceNormal, &InSurfaceMat, rng);
    int hit_index = -1;
    ShadeableIntersection _;
    getIntersectionGeometryIndex(debug,hit_index, _, wj, GeomSize, Geoms);
    if (hit_index != 0)
    {
       OutContribution = glm::vec3(0.f);
       return;
    }
    getGeomPDF(pdf_Ld, InLightGeom);
    float weight = power_heuristic(pdf_bsdf, pdf_Ld);
    OutContribution += directLight * bsdf * glm::max(0.f, glm::dot(wj.direction, InSurfaceNormal)) * weight / pdf_bsdf;    
}


__global__ void generateRayFromIntersections(int iter, int frame, int numPaths,
    PathSegment* pathSegments, ShadeableIntersection* dev_intersections,
    Scene::RenderProxy* scene,
    int* dev_pathAlive)
{
    int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
    int pathIndex = dev_pathAlive[tid];
    if (pathIndex < 0)
    {
        return;
    }
    if (tid >= numPaths)
    {
        return;
    }
    PathSegment path_segment = pathSegments[pathIndex];
    ShadeableIntersection intersection = dev_intersections[pathIndex];
    Geom light_geom = scene->geoms_Device[scene->light_index_Device[0]];
	Material light_mat = scene->materials_Device[light_geom.materialid];
    if (intersection.materialId < 0) {
        path_segment.remainingBounces = 0;
        pathSegments[pathIndex] = path_segment;
        return;
    }
    Material material = scene->materials_Device[intersection.materialId];
	glm::vec3 p = intersection.intersectPoint + EPSILON * intersection.surfaceNormal;
    thrust::default_random_engine rng = makeSeededRandomEngine(frame, pathIndex, iter * path_segment.remainingBounces);
    if (material.emittance > 0.)
    {
		path_segment.Contribution += path_segment.BSDF * material.emittance / path_segment.PDF * path_segment.Cosine;
        path_segment.remainingBounces = 0;
        pathSegments[pathIndex] = path_segment;
        return;
    }
    glm::vec3 contrib;
    glm::vec3 debug;
	glm::vec3 ViewDir = -path_segment.ray.direction;
    SampleDirectLightMIS(debug, contrib, p, ViewDir, intersection.surfaceNormal, material,
        light_geom, light_mat, scene->geoms_size, scene->geoms_Device, rng);
	path_segment.Contribution += path_segment.BSDF * contrib / path_segment.PDF * path_segment.Cosine;
    Ray wi;
    glm::vec3 bsdf_at_p;
	float pdf_bsdf;
    //bsdfDiffuseSample(bsdf_at_p, pdf_bsdf, wi, p, intersection.surfaceNormal, &material, rng);
	bsdfPBRSample(debug, bsdf_at_p, pdf_bsdf, wi, ViewDir, p, intersection.surfaceNormal, intersection.uv, material, rng);
	path_segment.BSDF *= bsdf_at_p;
    path_segment.PDF *= pdf_bsdf;
    if (path_segment.PDF<EPSILON)
    {
        path_segment.remainingBounces = 0;
        pathSegments[pathIndex] = path_segment;
		return;
    }
	path_segment.Cosine *= glm::max(0.f, glm::dot(wi.direction, intersection.surfaceNormal));

    path_segment.ray = wi;
	path_segment.remainingBounces--;
    pathSegments[pathIndex] = path_segment;
}