#include "hip/hip_runtime.h"
#include "pathtraceImpl.h"

#include "common.h"
#include "geometry.h"
#include "interactions.h"
#include "intersections.h"
#include "sceneStructs.h"
#include "utilities.h"
#include "mesh.h"


/**
* Generate PathSegments with rays from the camera through the screen into the
* scene, which is the first bounce of rays.
*
* Antialiasing - add rays for sub-pixel sampling
* motion blur - jitter rays "in time"
* lens effect - jitter ray origin positions based on a lens
*/
__global__ void generateRayFromCamera(Camera cam, int iter, int traceDepth, PathSegment* pathSegments, int* dev_pathAlive)
{
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    if (x < cam.resolution.x && y < cam.resolution.y) {
        int index = x + (y * cam.resolution.x);
        dev_pathAlive[index] = index;
        PathSegment& segment = pathSegments[index];
        thrust::default_random_engine rng = makeSeededRandomEngine(iter, index, traceDepth);
        thrust::uniform_real_distribution<float> u01(0, 1);
		float ramdomNumber1 = u01(rng);
		float ramdomNumber2 = u01(rng);
        segment.ray.origin = cam.position;
        segment.Contribution = glm::vec3(0.0f, 0.0f, 0.0f);
		segment.BSDF = glm::vec3(1.0f, 1.0f, 1.0f);
		segment.PDF = 1.0f;
        segment.Cosine = 1.0f;

        // TODO: implement antialiasing by jittering the ray
        segment.ray.direction = glm::normalize(cam.view
            - cam.right * cam.pixelLength.x * ((float)x + ramdomNumber1 - (float)cam.resolution.x * 0.5f)
            - cam.up * cam.pixelLength.y * ((float)y + ramdomNumber2 - (float)cam.resolution.y * 0.5f)
        );

        segment.pixelIndex = index;
        segment.remainingBounces = traceDepth;
    }
}

__device__ void getIntersectionGeometryIndex(
    float& t_min, int& hit_geom_index,
    glm::vec3& intersect_point,
    glm::vec3& normal,
    Ray& ray,
    int geoms_size,
	Geom* geoms, StaticMeshData_Device* dev_staticMeshes
)
{
    float t;
    t_min = FLT_MAX;
    bool outside = true;

    glm::vec3 tmp_intersect;
    glm::vec3 tmp_normal;

    // naive parse through global geoms

    for (int i = 0; i < geoms_size; i++)
    {
        Geom& geom = geoms[i];

        if (geom.type == CUBE)
        {
            t = boxIntersectionTest(geom, ray, tmp_intersect, tmp_normal, outside);
        }
        else if (geom.type == SPHERE)
        {
            t = sphereIntersectionTest(geom, ray, tmp_intersect, tmp_normal, outside);
        }else if (geom.type == PLANE)
        {
            t = planeIntersectionTest(geom, ray, tmp_intersect, tmp_normal, outside);
        }else if (geom.type==MESH && dev_staticMeshes!=nullptr)
        {
			//t = meshIntersectionTest(geom, dev_staticMeshes, ray, tmp_intersect, tmp_normal);
        }
        // TODO: add more intersection tests here... triangle? metaball? CSG?

        // Compute the minimum t from the intersection tests to determine what
        // scene geometry object was hit first.
        if (t > 0.0f && t_min > t)
        {
            t_min = t;
            hit_geom_index = i;
            intersect_point = tmp_intersect;
            normal = tmp_normal;
        }
    }
}

__global__ void computeIntersections(
    int depth,
    int num_paths,
    PathSegment* pathSegments,
    Geom* geoms,
    int geoms_size,
    ShadeableIntersection* intersections,
    int* device_materialIds, int* dev_pathAlive, StaticMeshData_Device* dev_staticMeshes)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int path_index = dev_pathAlive[tid];
    if (path_index < 0 || tid >= num_paths)
    {
        return;
    }
    PathSegment pathSegment = pathSegments[path_index];
	// If there are no remaining bounces, no need to trace
    if (pathSegment.remainingBounces <= 0) {
        intersections[path_index].t = -1.0f;
        intersections[path_index].materialId = -1;
        device_materialIds[path_index] = -1;
        dev_pathAlive[tid] = -1;
        return;
	}
    int hit_geom_index = -1;
	float t_min = FLT_MAX;
    glm::vec3 intersect_point;
    glm::vec3 normal;

    getIntersectionGeometryIndex(
        t_min, hit_geom_index, intersect_point, normal,
        pathSegment.ray, geoms_size, geoms, dev_staticMeshes);

    if (hit_geom_index == -1)
    {
        intersections[path_index].t = -1.0f;
        intersections[path_index].materialId = -1;
        device_materialIds[path_index] = -1;
        dev_pathAlive[tid] = -1;
        //pathSegment.remainingBounces = 0;
        //pathSegments[path_index] = pathSegment;
        pathSegments[path_index].Contribution = glm::vec3(0.f, 0.f, 1.f);
    }
    else
    {
        // The ray hits something
        int matId = geoms[hit_geom_index].materialid;
        intersections[path_index].t = t_min;
        intersections[path_index].materialId = matId;
        device_materialIds[path_index] = matId;
        intersections[path_index].surfaceNormal = normal;
		pathSegments[path_index].Contribution = glm::vec3(1.f, 1.f, 0.f);
    }
}

__host__ __device__ float power_heuristic(float pdf_a, float pdf_b) {
    float a = pdf_a * pdf_a;
    float b = pdf_b * pdf_b;
    return a / (a + b);
}

__device__ void bsdfDiffuse(glm::vec3& outBSDF, float& outPDF, const Ray& in_wi, glm::vec3 surfaceNormal, Material* material)
{
    float absdot = glm::abs(glm::dot(in_wi.direction, surfaceNormal));
    float pdf = absdot * INV_PI;
	outPDF = pdf;
    outBSDF = material->color * INV_PI;
}

__device__ void bsdfDiffuseSample(glm::vec3& outBSDF, float& outPDF, Ray& out_wi, glm::vec3 p,
    glm::vec3 surfaceNormal, Material* material,
    thrust::default_random_engine& rng)
{
    // set segment.ray.origin
    out_wi.origin = p;
    // set segment.ray.direction on random position
    out_wi.direction = calculateRandomDirectionInHemisphere(surfaceNormal, rng);
    bsdfDiffuse(outBSDF, outPDF, out_wi, surfaceNormal, material);
}

__device__ void bsdfSpecular(PathSegment* wi, PathSegment* wo, Material* material)
{

}

__device__ void bsdfEmitting(PathSegment* wo, Material* material)
{
    //wo->remainingBounces = 0;
    //wo->color *= material->emittance;
}

__device__ void getGeomPDF(float& outPdf, Geom& InGeom)
{
    if (InGeom.type == CUBE)
    {
        //pdfCube(outPdf, InGeom);
    }
    else if (InGeom.type == PLANE)
    {
        pdfPlane(outPdf, InGeom);
    }
}

__device__ bool sampleLightFromIntersections(
    glm::vec3& outDirectLight,
    float& outPdf,
    Ray& wj,
	glm::vec3 p, // intersection point
    const Material& light_mat,
    Geom& light_geom,
    int geomSize,
	Geom* geoms,
    thrust::default_random_engine& rng
)
{
    glm::vec3 lightPosition;
    glm::vec3 lightNormal;
	outPdf = 1.0f;
    sampleGeometry(light_geom, lightPosition, lightNormal, outPdf, rng);
    wj.origin = p;
	wj.direction = glm::normalize(lightPosition - wj.origin);
    float distance = glm::length(lightPosition - wj.origin);
    // convert pdf from area to solid angle
	// pdf_L(direct light) = pdf_A * (distance * distance) / (n . wj)
	float dotProduct = glm::dot(lightNormal, wj.direction);
    outPdf /= dotProduct;
    outPdf *= (distance * distance);
	float t;
	glm::vec3 tmp_intersect;
	glm::vec3 tmp_normal;
	int hit_geom_index = -1;
	getIntersectionGeometryIndex(
		t, hit_geom_index,
        tmp_intersect, tmp_normal,
        wj, geomSize, geoms, nullptr);
    outDirectLight += light_mat.emittance;
    return dotProduct > 0.0001f && glm::length(tmp_intersect - lightPosition) < 0.001f;
}

__device__ void SampleDirectLightMIS(glm::vec3& OutContribution, 
    glm::vec3 In_p, glm::vec3 InSurfaceNormal, Material& InSurfaceMat, 
    Geom& InLightGeom, Material& InLightMat, int GeomSize, Geom* Geoms,
    thrust::default_random_engine& rng)
{
    glm::vec3 directLight;
    float pdf_Ld;
    glm::vec3 bsdf;
    float pdf_bsdf;
    Ray wj;
    bool sampledDirectLight = sampleLightFromIntersections(directLight, pdf_Ld, wj, In_p, InLightMat, InLightGeom, GeomSize, Geoms, rng);
    if (sampledDirectLight)
    {
        bsdfDiffuse(bsdf, pdf_bsdf, wj, InSurfaceNormal, &InSurfaceMat);
        float weight = power_heuristic(pdf_Ld, pdf_bsdf);
        OutContribution += directLight * bsdf * glm::max(0.f, glm::dot(wj.direction, InSurfaceNormal)) * weight / pdf_Ld;
    }
    bsdfDiffuseSample(bsdf, pdf_bsdf, wj, In_p, InSurfaceNormal, &InSurfaceMat, rng);
    float t;
    int hit_index;
    glm::vec3 intersect, normal;
    getIntersectionGeometryIndex(t, hit_index, intersect, normal, wj, GeomSize, Geoms, nullptr);
    if (hit_index == 0)
    {
        getGeomPDF(pdf_Ld, InLightGeom);
        float weight = power_heuristic(pdf_bsdf, pdf_Ld);
        OutContribution += directLight * bsdf * glm::max(0.f, glm::dot(wj.direction, InSurfaceNormal)) * weight / pdf_bsdf;
    }
    if (!sampledDirectLight || hit_index != 0)
    {
        OutContribution = glm::vec3(0.f);
    }
}


__global__ void generateRayFromIntersections(int iter, int numPaths,
    PathSegment* pathSegments, ShadeableIntersection* dev_intersections,
    Material* inMaterial, int geomSize, Geom* geoms, Geom* light_geoms, int* dev_pathAlive)
{
    int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
    int pathIndex = dev_pathAlive[tid];
    if (pathIndex < 0)
    {
        return;
    }
    if (tid >= numPaths)
    {
        return;
    }
    PathSegment path_segment = pathSegments[pathIndex];
    ShadeableIntersection intersection = dev_intersections[pathIndex];
	Material light_mat = inMaterial[light_geoms[0].materialid];
	Geom light_geom = light_geoms[0];
    if (intersection.materialId < 0) {
        path_segment.remainingBounces = 0;
        pathSegments[pathIndex] = path_segment;
        return;
    }

    if (intersection.t > 0.0f && path_segment.remainingBounces > 0) {
        Material material = inMaterial[intersection.materialId];
		glm::vec3 p = getPointOnRay(path_segment.ray, intersection.t);
        thrust::default_random_engine rng = makeSeededRandomEngine(iter, pathIndex, path_segment.remainingBounces);
        if (material.emittance > 0.)
        {
			path_segment.Contribution += path_segment.BSDF * material.emittance / path_segment.PDF * path_segment.Cosine;
            path_segment.remainingBounces = 0;
            pathSegments[pathIndex] = path_segment;
            return;
        }
        glm::vec3 contrib;
        SampleDirectLightMIS(contrib, p, intersection.surfaceNormal, material, 
            light_geom, light_mat, geomSize, geoms, rng);
		path_segment.Contribution += path_segment.BSDF * contrib / path_segment.PDF * path_segment.Cosine;
        Ray wi;
        glm::vec3 bsdf_at_p;
		float pdf_bsdf;
        bsdfDiffuseSample(bsdf_at_p, pdf_bsdf, wi, p, intersection.surfaceNormal, &material, rng);
		path_segment.BSDF *= bsdf_at_p;
        path_segment.PDF *= pdf_bsdf;
		path_segment.Cosine *= glm::max(0.f, glm::dot(wi.direction, intersection.surfaceNormal));

        path_segment.ray = wi;
		path_segment.remainingBounces--;
    	pathSegments[pathIndex] = path_segment;
    }
}